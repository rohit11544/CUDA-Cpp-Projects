#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <functional>
#include <vector>
#include <cassert>
#include "Header.h"

using namespace std;

main() {
	const int N = 1 << 10;
	size_t bytes = N * N * sizeof(int);

	vector<int> h_a(N * N);
	vector<int> h_b(N * N);
	vector<int> h_c(N * N);

	generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
	generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

	int* d_a, * d_b, * d_c;

	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

	int THREADS = 32;
	int BLOCKS = N / THREADS;

	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);

	baseLine << <blocks, threads >> > (d_a, d_b, d_c, N);
	hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);
	verify_result(h_a, h_b, h_c, N);
	cout << "COMPLETED BASELINE SUCCESSFULLY\n";
	
	TiledMatrixMul << <blocks, threads >> > (d_a, d_b, d_c);
	hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);
	verify_result(h_a, h_b, h_c, N);
	cout << "COMPLETED TILEING SUCCESSFULLY\n";

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}