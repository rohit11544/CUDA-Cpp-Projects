#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#define MASK_LENGTH 7

__constant__ int mask[MASK_LENGTH];

__global__ void convolution_1d(int* array, int* result, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ int s_array[];

    int r = MASK_LENGTH / 2;

    int d = 2 * r;

    int n_padded = blockDim.x + d;

    int offset = threadIdx.x + blockDim.x;

    int g_offset = blockDim.x * blockIdx.x + offset;

    s_array[threadIdx.x] = array[tid];

    if (offset < n_padded) {
        s_array[offset] = array[g_offset];
    }
    __syncthreads();

    int temp = 0;

    for (int j = 0; j < MASK_LENGTH; j++) {
        temp += s_array[threadIdx.x + j] * mask[j];
    }

    result[tid] = temp;
}

void verify_result(int* array, int* mask, int* result, int n) {
    int temp;
    for (int i = 0; i < n; i++) {
        temp = 0;
        for (int j = 0; j < MASK_LENGTH; j++) {
            temp += array[i + j] * mask[j];
        }
        assert(temp == result[i]);
    }
}

int main() {
    int n = 1 << 20;

    int bytes_n = n * sizeof(int);

    size_t bytes_m = MASK_LENGTH * sizeof(int);

    int r = MASK_LENGTH / 2;
    int n_p = n + r * 2;

    size_t bytes_p = n_p * sizeof(int);

    int* h_array = new int[n_p];

    for (int i = 0; i < n_p; i++) {
        if ((i < r) || (i >= (n + r))) {
            h_array[i] = 0;
        }
        else {
            h_array[i] = rand() % 100;
        }
    }

    int* h_mask = new int[MASK_LENGTH];
    for (int i = 0; i < MASK_LENGTH; i++) {
        h_mask[i] = rand() % 10;
    }

    int* h_result = new int[n];

    int* d_array, * d_result;
    hipMalloc(&d_array, bytes_p);
    hipMalloc(&d_result, bytes_n);

    hipMemcpy(d_array, h_array, bytes_p, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

    int THREADS = 256;

    int GRID = (n + THREADS - 1) / THREADS;

    size_t SHMEM = (THREADS + r * 2) * sizeof(int);

    convolution_1d << <GRID, THREADS, SHMEM >> > (d_array, d_result, n);

    hipMemcpy(h_result, d_result, bytes_n, hipMemcpyDeviceToHost);

    verify_result(h_array, h_mask, h_result, n);

    std::cout << "COMPLETED SUCCESSFULLY\n";

    delete[] h_array;
    delete[] h_result;
    delete[] h_mask;
    hipFree(d_result);

    return 0;
}