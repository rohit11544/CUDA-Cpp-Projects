#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include "Header.h"


#define MASK_LENGTH 7

__constant__ int mask[MASK_LENGTH];


int main() {
	const int N = 1 << 20, M = 7;
	size_t bytes = N * sizeof(int);

	int* h_a, * h_m, * h_r;
	h_a = new int[N];
	h_r = new int[N];
	h_m = new int[M];

	for (int i = 0; i < N; i++) {
		h_a[i] = rand() % 100;
	}
	
	for (int i = 0; i < M; i++) {
		h_m[i] = rand() % 10;
	}

	int* d_a, * d_m, * d_r;

	hipMalloc(&d_a, bytes);
	hipMalloc(&d_m, M * sizeof(int));
	hipMalloc(&d_r, bytes);

	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_m, h_m, M*sizeof(int), hipMemcpyHostToDevice);

	int thread_size = 256;
	int grid_size = (N + thread_size - 1) / thread_size;

	BaseLine << <grid_size, thread_size >> > (d_a, d_m, d_r, N, M);

	hipMemcpy(h_r, d_r, bytes, hipMemcpyDeviceToHost);

	verify_result(h_a, h_m, h_r, M, N);

	std::cout << "COMPLETED SUCCESSFUL" << std::endl;
	
	int* d_r_2;
	hipMalloc(&d_r_2, M * sizeof(int));
	
	hipMemcpyToSymbol(HIP_SYMBOL(mask), h_r, M * sizeof(int));

	ConstantMem << <grid_size, thread_size >> > (d_a, d_r_2, N);

	hipMemcpy(h_r, d_r_2, bytes, hipMemcpyDeviceToHost);

	verify_result(h_a, h_m, h_r, M, N);

	std::cout << "COMPLETED SUCCESSFUL" << std::endl;
	

	hipFree(d_a);
	hipFree(d_r);
	hipFree(d_m);
	hipFree(d_r_2);

	delete[] h_a;
	delete[] h_r;
	delete[] h_m;
	 
	return 0;
}

