#include "hip/hip_runtime.h"
#include "Header.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
using namespace std;

#define SHMEM_SIZE 256

__global__ void LaunchHalfThreads(int* a, int* a_v) {
	__shared__ int partial_sum[SHMEM_SIZE];

	int i = 2 * blockDim.x * blockIdx.x + threadIdx.x;
	partial_sum[threadIdx.x] = a[i] + a[i + blockDim.x];

	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		a_v[blockIdx.x] = partial_sum[0];
	}

}