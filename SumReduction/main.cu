#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "Header.h"
using namespace std;

int main() {
	
	const int N = 1 << 16;
	size_t bytes = N * sizeof(int);

	int* h_a, * h_a_v;

	h_a = new int[N];
	h_a_v = new int[N];
	
	generateArray(h_a, N);

	int* d_a, * d_a_v;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_a_v, bytes);
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

	const int TB_SIZE = 256;
	int GRID_SIZE = N / TB_SIZE;

	BaseLine << < GRID_SIZE, TB_SIZE >> > (d_a, d_a_v);
	
	BaseLine << < 1, TB_SIZE >> > (d_a_v, d_a_v);
	
	hipMemcpy(h_a_v, d_a_v, bytes, hipMemcpyDeviceToHost);

	verifyResult(h_a, h_a_v, N);
	
	cout << "COMPLETED BaseLine SUCCESSFULLY" << endl;
	
	int* d_a_2, * d_a_v_2;
	hipMalloc(&d_a_2, bytes);
	hipMalloc(&d_a_v_2, bytes);
	hipMemcpy(d_a_2, h_a, bytes, hipMemcpyHostToDevice);
	
	NoWarpDiversion << < GRID_SIZE , TB_SIZE >> > (d_a_2, d_a_v_2);

	NoWarpDiversion << < 1, TB_SIZE >> > (d_a_v_2, d_a_v_2);

	hipMemcpy(h_a_v, d_a_v_2, bytes, hipMemcpyDeviceToHost);

	verifyResult(h_a, h_a_v, N);
	
	cout << "COMPLETED NoWarpDiversion SUCCESSFULLY" << endl;

	int* d_a_3, * d_a_v_3;
	hipMalloc(&d_a_3, bytes);
	hipMalloc(&d_a_v_3, bytes);
	hipMemcpy(d_a_3, h_a, bytes, hipMemcpyHostToDevice);

	NoBankConflits << < GRID_SIZE , TB_SIZE >> > (d_a_3, d_a_v_3);

	NoBankConflits << < 1, TB_SIZE >> > (d_a_v_3, d_a_v_3);

	hipMemcpy(h_a_v, d_a_v_3, bytes, hipMemcpyDeviceToHost);

	verifyResult(h_a, h_a_v, N);

	cout << "COMPLETED NoBankConflits SUCCESSFULLY" << endl;

	int* d_a_4, * d_a_v_4;
	hipMalloc(&d_a_4, bytes);
	hipMalloc(&d_a_v_4, bytes);
	hipMemcpy(d_a_4, h_a, bytes, hipMemcpyHostToDevice);

	LaunchHalfThreads << < GRID_SIZE / 2, TB_SIZE >> > (d_a_4, d_a_v_4);

	LaunchHalfThreads << < 1, TB_SIZE >> > (d_a_v_4, d_a_v_4);

	hipMemcpy(h_a_v, d_a_v_4, bytes, hipMemcpyDeviceToHost);

	verifyResult(h_a, h_a_v, N);

	cout << "COMPLETED LaunchHalfThreads SUCCESSFULLY" << endl;


	delete[] h_a;
	delete[] h_a_v;
	
	hipFree(d_a);
	hipFree(d_a_v);
	hipFree(d_a_2);
	hipFree(d_a_v_2);
	hipFree(d_a_3);
	hipFree(d_a_v_3);
	hipFree(d_a_4);
	hipFree(d_a_v_4);



	return 0;

}