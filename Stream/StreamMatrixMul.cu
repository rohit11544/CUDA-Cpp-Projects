#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <functional>
#include <vector>
#include <cassert>

using namespace std;

__global__ void baseLine(const int* a, const int* b, int* c, int N) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	c[row * N + col] = 0;

	for (int i = 0; i < N; i++) {
		c[row * N + col] += a[row * N + i] * b[col + i * N];
	}
}

int verify_result(vector<int>& a, vector<int>& b, vector<int>& c, int N) {

	for (int i = 0; i < N; i++) {

		int flag = 0;
		for (int j = 0; j < N; j++) {

			int tmp = 0;
			for (int k = 0; k < N; k++) {

				tmp += a[i * N + k] * b[k * N + j];
			}

			if (tmp == c[i * N + j]) { flag = 1; }

			if (flag == 0) {
				cout << "Wronge" << endl;
				exit(0);
			}
		}
	}

	cout << "COMPLETED SUCCESSFULLY\n";
}


main() {
	const int N = 1 << 7;
	size_t bytes = N * N * sizeof(int);

	vector<int> h_1(N * N), h_2(N * N), h_r1(N * N);
	vector<int> h_3(N * N), h_4(N * N), h_r2(N * N);
	vector<int> h_5(N * N), h_6(N * N), h_r3(N * N);

	generate(h_1.begin(), h_1.end(), []() { return rand() % 100; });
	generate(h_2.begin(), h_2.end(), []() { return rand() % 100; });

	generate(h_3.begin(), h_3.end(), []() { return rand() % 100; });
	generate(h_4.begin(), h_4.end(), []() { return rand() % 100; });

	generate(h_5.begin(), h_5.end(), []() { return rand() % 100; });
	generate(h_6.begin(), h_6.end(), []() { return rand() % 100; });

	int* d_1, * d_2, * d_r1;
	int* d_3, * d_4, * d_r2;
	int* d_5, * d_6, * d_r3;

	hipMalloc(&d_1, bytes);
	hipMalloc(&d_2, bytes);
	hipMalloc(&d_r1, bytes);

	hipMalloc(&d_3, bytes);
	hipMalloc(&d_4, bytes);
	hipMalloc(&d_r2, bytes);

	hipMalloc(&d_5, bytes);
	hipMalloc(&d_6, bytes);
	hipMalloc(&d_r3, bytes);

	int THREADS = 32;
	int BLOCKS = N / THREADS;

	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);

	hipStream_t stream[3];
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);
	hipStreamCreate(&stream[2]);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// start recording
	hipEventRecord(start);

	hipMemcpyAsync(d_1, h_1.data(), bytes, hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(d_3, h_3.data(), bytes, hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(d_5, h_5.data(), bytes, hipMemcpyHostToDevice, stream[2]);

	hipMemcpyAsync(d_2, h_2.data(), bytes, hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(d_4, h_4.data(), bytes, hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(d_6, h_6.data(), bytes, hipMemcpyHostToDevice, stream[2]);

	baseLine << <blocks, threads, 0, stream[0] >> > (d_1, d_2, d_r1, N);
	baseLine << <blocks, threads, 0, stream[1] >> > (d_3, d_4, d_r2, N);
	baseLine << <blocks, threads, 0, stream[2] >> > (d_5, d_6, d_r3, N);

	hipMemcpyAsync(h_r1.data(), d_r1, bytes, hipMemcpyDeviceToHost, stream[0]);
	hipMemcpyAsync(h_r2.data(), d_r2, bytes, hipMemcpyDeviceToHost, stream[1]);
	hipMemcpyAsync(h_r3.data(), d_r3, bytes, hipMemcpyDeviceToHost, stream[2]);

	//stop recording
	hipEventRecord(stop);

	verify_result(h_1, h_2, h_r1, N);
	verify_result(h_3, h_4, h_r2, N);
	verify_result(h_5, h_6, h_r3, N);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "Total time in ms taken to complete kernels : " << milliseconds << endl;

	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
	hipStreamDestroy(stream[2]);

	hipFree(d_1);
	hipFree(d_2);
	hipFree(d_3);
	hipFree(d_4);
	hipFree(d_5);
	hipFree(d_6);
	hipFree(d_r1);
	hipFree(d_r2);
	hipFree(d_r3);

	return 0;
}